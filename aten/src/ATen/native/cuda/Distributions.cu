#include "ATen/NativeFunctions.h"
#include "ATen/Dispatch.h"
#include "ATen/cuda/CUDAApplyUtils.cuh"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <THC/THCTensorMath.h>
#include <THC/THCGeneral.h>
#include <THC/THCHalf.h>
#include <THC/THCTensorCopy.h>
#include <THC/THCApply.cuh>
#include <THC/THCNumerics.cuh>
#include <THC/THCReduce.cuh>
#include <THC/THCTensorRandom.h>

#include <THC/THCDeviceUtils.cuh>
#include <THC/THCTensorMathReduce.cuh>
#include <THC/THCTensorSort.cuh>
#include <THC/THCThrustAllocator.cuh>


THCGenerator* THCRandom_getGenerator(THCState* state);

namespace at {
namespace native {

namespace dist {
  hiprandStateMtgp32* get_states(Generator *gen) {
    auto gen_ = THCRandom_getGenerator(at::globalContext().thc_state);
    return gen_->gen_states;
  }
  
  template <typename scalar>
  struct PoissonOpCUDA {
    static void apply(Tensor& ret, const Tensor& lambda, hiprandStateMtgp32 *states) {
      at::cuda::CUDA_tensor_apply2<scalar, float>(ret, lambda,
        [states] __device__ (scalar& ret_val, const float& lambda, bool early_exit) {
          ret_val = scalar_cast<scalar>(hiprand_poisson(&states[blockIdx.x], lambda));
        }
      );
    }
  };
} // at::native::dist

Tensor _s_poisson_cuda(const Tensor& lambda, Generator* gen) {
  Tensor ret = lambda.type().tensor(lambda.sizes());
  auto lambda_ = lambda.toType(ScalarType::Float);
  dispatch_all<void, dist::PoissonOpCUDA>(ret.type(), "poisson", ret, lambda_, dist::get_states(gen));
  return ret;
}

} // at::native
} // at
