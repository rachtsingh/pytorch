#include "hip/hip_runtime.h"
#include "ATen/NativeFunctions.h"
#include "ATen/Dispatch.h"
#include "ATen/cuda/CUDAApplyUtils.cuh"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_kernel.h>
#include <utility>

#include <THC/THCGeneral.h>
#include <THC/THCHalf.h>
#include <THC/THCApply.cuh>
#include <THC/THCNumerics.cuh>
#include <THC/THCTensorRandom.h>

THCGenerator* THCRandom_getGenerator(THCState* state);

namespace at {
namespace native {

namespace dist {
  std::pair<uint64_t, uint64_t> get_philox_seed(Generator *gen) {
    auto gen_ = THCRandom_getGenerator(at::globalContext().thc_state);
    return std::make_pair(gen_->initial_seed, gen_->philox_seed_offset++);
  }

  // note that sample_poisson is adapted from Numpy's distributions.c
  // see Distributions.cpp for the license
  __device__ int64_t sample_poisson(float lambda, hiprandStatePhilox4_32_10_t *state) {
    if (lambda >= 10) {
      // transformed rejection method, (Hoermann, 1993)
      int64_t k;
      float U, V, a, b, invalpha, vr, us;

      float slam = ::sqrt(lambda);
      float loglam = ::log(lambda);
      b = 0.931 + 2.53 * slam;
      a = -0.059 + 0.02483 * b;
      invalpha = 1.1239 + 1.1328/(b-3.4);
      vr = 0.9277 - 3.6224/(b-2);

      while (1) {
        U = hiprand_uniform(state) - 0.5;
        V = hiprand_uniform(state);
        us = 0.5 - ::fabs(U);
        k = (int64_t) ::floor((2*a/us + b)*U + lambda + 0.43);
        if ((us >= 0.07) && (V <= vr)) {
          return k;
        }
        if ((k < 0) || ((us < 0.013) && (V > us))) {
          continue;
        }
        if ((::log(V) + ::log(invalpha) - ::log(a/(us*us)+b)) <= (-lambda + k*loglam - ::lgamma((float) k+1)))
        {
          return k;
        }
      }
    }
    else if (lambda == 0) {
      return 0;
    }
    else {
      int64_t X;
      float prod, U, enlam;

      enlam = ::exp(-lambda);
      X = 0;
      prod = 1.0;
      while (1) {
        U = hiprand_uniform(state);
        prod *= U;
        if (prod > enlam) {
          X += 1;
        }
        else {
          return X;
        }
      }
    }
  }
  
  template <typename scalar>
  struct PoissonOpCUDA {
    static void apply(Tensor& ret, const Tensor& lambda, std::pair<uint64_t, uint64_t> seeds) {
      at::cuda::CUDA_tensor_apply2<scalar, float>(ret, lambda,
        [seeds] __device__ (scalar& ret_val, const float& lambda, bool early_exit) {
          hiprandStatePhilox4_32_10_t state;
          hiprand_init(seeds.first, blockIdx.x * blockDim.x + threadIdx.x, seeds.second, &state);
          ret_val = scalar_cast<scalar>(sample_poisson(lambda, &state));
        }
      );
    }
  };

} // at::native::dist

Tensor _s_poisson_cuda(const Tensor& lambda, Generator* gen) {
  Tensor ret = lambda.type().tensor(lambda.sizes());
  auto lambda_ = lambda.toType(ScalarType::Float);
  dispatch_floating_types<void, dist::PoissonOpCUDA>(ret.type(), "poisson", ret, lambda_, dist::get_philox_seed(gen));
  return ret;
}

} // at::native
} // at
